#include "hip/hip_runtime.h"
#include "Kernel.h"

__global__ void initCurand(hiprandStatePhilox4_32_10_t* states, const glm::uvec2 imgDim, const size_t seed)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t gIndex = x + y * blockDim.x * gridDim.x;

	if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= gIndex) {
		return;
	}
	hiprand_init(seed, (size_t)gIndex, 0, &states[gIndex]);
	// Sequence 0 and offset 0 for better performance but may result in worse 'randomness'
}

__device__ glm::vec3 randomDirectionUnitSphere(hiprandStatePhilox4_32_10_t* rndState) 
{
	auto rndVec3 = [&rndState]() -> glm::vec3 {
		return glm::vec3(2.f * hiprand_uniform(rndState) - 1.f);
	};
	glm::vec3 randomPoint = rndVec3();
	while (1.f <= glm::length(randomPoint)) {
		randomPoint = rndVec3();
	}
	return randomPoint;
}

__device__ HitRecord traceRay(const Ray ray, const Sphere* hittable, const uint32_t hittableSize)
{
	int closestObjIdx = -1;
	float closestT{ FLT_MAX };
	glm::vec3 shiftOrigin{};

	for (int i = 0; i < hittableSize; i++) {
		// Shifing current camera to the position of given object. It's used for the calculation of intersections.
		shiftOrigin = ray.origin - hittable[i].getPosition();
		float t = hittable[i].hit({ shiftOrigin, ray.direction });
		if (t < 0.f)
			continue;

		if (t < closestT) {
			closestObjIdx = i;
			closestT = t;
		}
	}

	if (closestObjIdx < 0) {
		return miss(ray);
	}
	return closestHit(ray, closestT, closestObjIdx, hittable);
}


__device__ HitRecord closestHit(const Ray ray, float hitDistance, int objectIndex, const Sphere* hittable)
{
	const Sphere& closestSphere = hittable[objectIndex];
	glm::vec3 origin = ray.origin - closestSphere.getPosition(); // Move back to the origin
	
	glm::vec3 hitPoint = origin + ray.direction * hitDistance;
	glm::vec3 normal = glm::normalize(hitPoint);

	hitPoint += closestSphere.getPosition(); // Move into real position

	HitRecord hitRecord(ray.direction, normal, hitPoint, hitDistance, objectIndex);
	return hitRecord;
}


__device__ HitRecord miss(const Ray ray)
{
	return HitRecord();
}

//template <int DEPTH>
//__device__ glm::vec4 colorRaw(const Ray ray, const Sphere* hittable, const uint32_t hittableSize, const glm::vec4& backgroundColor, hiprandStatePhilox4_32_10_t* rndState)
//{
//	const Sphere* closestSphere = nullptr;
//	Ray nRay = ray;
//	float color{ 1.f };
//	float lightIntensity{1.f};
//	int n = DEPTH;
//	do {
//		glm::vec3 closestShiftOrigin{};
//		float closestT{ FLT_MAX };
//		for (int i = 0; i < hittableSize; i++) {
//			// Shifing current camera to the position of given object. It's used for the calculation of intersections.
//			glm::vec3 shiftOrigin = nRay.origin - hittable[i].getPosition();
//			float t = hittable[i].hit({ shiftOrigin, nRay.direction });
//			if (t < 0.f)
//				continue;
//
//			if (t < closestT) {
//				closestSphere = &hittable[i];
//				closestT = t;
//				closestShiftOrigin = shiftOrigin;
//			}
//		}
//
//		if (closestSphere == nullptr) {
//			return color * backgroundColor;
//		}
//
//		glm::vec3 closestHit = closestT * nRay.direction + closestShiftOrigin;
//
//		//HitRecord hitRecord(nRay.direction, (closestHit - closestSphere->getPosition()) / closestSphere->getRadius()); // normal as unit vector of closestHit so the light is global
//
//		color *= 0.5f;
//		nRay.origin = closestHit;
//		glm::vec3 target = hitRecord.normal + randomDirection(rndState, closestHit);
//		nRay.direction = target;
//		closestSphere = nullptr;
//
//		//glm::vec3 lightSource = glm::normalize(glm::vec3(1.f, 1.f, -1.f));
//		//lightIntensity = glm::max(glm::dot(closestHit, -lightSource), 0.f); // only angles: 0 <= d <= 90
//
//	} while (0 < n--);
//
//	return glm::vec4(0.f, 0.f, 0.f, 1.f);
//	/*return {
//			color.r * lightIntensity,
//			color.g * lightIntensity,
//			color.b * lightIntensity,
//			color.a
//	};*/
//}

__global__ void perPixel(
	uint32_t* imgBuff,
	glm::vec4* accColor,
	const glm::uvec2 imgDim,
	hiprandStatePhilox4_32_10_t* rndState,
	const Sphere* hittable,
	const uint32_t hittableSize,
	const Material* material,
	const Camera* camera,
	const uint32_t accN) {

	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t gIndex = x + y * blockDim.x * gridDim.x;

	if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= gIndex) {
		return;
	}
	glm::vec2 coord = {((float)x * 2.f / (float)imgDim.x) - 1.f,
						((float)y * 2.f / (float)imgDim.y) - 1.f}; // [-1; 1]

	float grad = 0.5f * (-coord.y + 1.f);
	glm::vec4 backgroundColor = {(1.f - grad) * glm::vec3(1.f, 1.f, 1.f) + grad * glm::vec3(0.4f, 0.6f, 0.8f), 1.f};
	// backgroundColor = glm::vec4(0.f, 0.f, 0.f, 1.f);

	if (!hittableSize) {
		imgBuff[gIndex] = convertFromRGBA(backgroundColor);
		return;
	}

	const int BOUNCES = 30;
	Ray ray;
	HitRecord hitRecord;
	glm::vec3 lightSource = glm::normalize(glm::vec3(-1.f, -1.f, -1.f));
	glm::vec4 light{0.0f, 0.0f, 0.0f, 1.f};

	ray.origin = camera->GetPosition();
	
	for (int i = 0; i < ANTIALIASING_SAMPLES; i++) {
		glm::vec2 rndCoord{
			( (x + hiprand_uniform(&rndState[gIndex])) * 2.f ) / float(imgDim.x) - 1.f,
			( (y + hiprand_uniform(&rndState[gIndex])) * 2.f ) / float(imgDim.y) - 1.f};

		ray.direction = camera->calculateRayDirection(coord);
		glm::vec3 contribution = { 1.0f, 1.0f, 1.0f};
		for (int j = 0; j < BOUNCES; j++){
			hitRecord = traceRay(ray, hittable, hittableSize);
			const Sphere* sphere = &hittable[hitRecord.objectIndex];
			const Material* mat = &material[sphere->getMaterialIdx()];

			if (hitRecord.distance < 0.f) { // Didn't hit any hittable
				light += backgroundColor * glm::vec4(contribution, 1.0f);
				break;
			}
			else {
				float lightIntensity = glm::max(glm::dot(hitRecord.normal, -lightSource), 0.f); // only angles: 0 <= d <= 90
				
				contribution *= glm::vec3(mat->color.r, mat->color.g, mat->color.b);
				//light += material->color * glm::vec4(contribution, 1.0f) * lightIntensity; // light intensity might be optional
				//light += material->color * glm::vec4(contribution, 1.0f) * glm::vec4(material->getEmmision(), 1.0f) * lightIntensity;
				light +=  glm::vec4(mat->getEmmision(), 1.0f);

			}
			ray.origin = hitRecord.position + hitRecord.normal * 0.0001f;
			ray.direction = glm::reflect(glm::normalize(ray.direction), hitRecord.normal) + mat->roughness * (randomDirectionUnitSphere(&rndState[gIndex]));
			// ray.direction = hitRecord.normal + randomDirectionUnitSphere(&rndState[gIndex]);
		}
	}
	glm::vec4 color = light / (float)ANTIALIASING_SAMPLES;

	uint32_t& buff = imgBuff[gIndex];
	glm::vec4& acc = accColor[gIndex];

	glm::vec4 currAcc{acc};

	if (accN <= 1) {
		acc = glm::vec4(
			color.r,
			color.g,
			color.b,
			1.f);
	}
	else {
		acc += glm::vec4(
			color.r,
			color.g,
			color.b,
			1.f);
		currAcc = acc / glm::vec4(accN);
	}
	buff = convertFromRGBA(currAcc);
}


Kernel::Kernel(): kernelTimeMs(0.f), TPB(16){
}

void Kernel::runKernel(const Scene& scene, const Camera& camera, const Settings settings) {
	// TODO: Je�li to b�dzie w p�tli si� od�wie�a�o to warto nie alokowa� tego za ka�dym razem
	uint32_t* d_buffer = nullptr;
	glm::vec4* d_accColor = nullptr;
	Sphere* d_hittable = nullptr;
	Material* d_material = nullptr;
	Camera* d_camera = nullptr;
	hiprandStatePhilox4_32_10_t* d_curandState = nullptr;
	hipEvent_t start, stop;
	uint32_t bufferSize = imgDim.x * imgDim.y;
	dim3 gridDim((imgDim.x + TPB - 1) / TPB, (imgDim.y + TPB - 1) / TPB);
	dim3 blockDim(TPB, TPB);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    if (!bufferSize) {
		throw std::invalid_argument("CUDA: buffer size is not set!");
    } 
	else if (!buffer) {
		throw std::invalid_argument("CUDA: buffer is NULL!");
    }
	else if (!accColor) {
		throw std::invalid_argument("CUDA: accColor buffer is NULL!");
	}

	gpuErrChk(hipMalloc(&d_buffer,  bufferSize * sizeof(*d_buffer)));
	gpuErrChk(hipMalloc(&d_accColor, bufferSize * sizeof(*d_accColor)));

	gpuErrChk(hipMalloc(&d_hittable, scene.sphere.size() * sizeof(*d_hittable)));
	gpuErrChk(hipMalloc(&d_material, scene.material.size() * sizeof(*d_material)));
	gpuErrChk(hipMalloc(&d_curandState, bufferSize * sizeof(*d_curandState)));
	gpuErrChk(hipMalloc(&d_camera, sizeof(*d_camera)));
	hipEventRecord(start);

	auto duration = std::chrono::system_clock::now().time_since_epoch();
	initCurand << < gridDim, blockDim >> > (d_curandState, imgDim, size_t(duration.count()));

    gpuErrChk(hipMemcpy(d_hittable, scene.sphere.data(),
                         scene.sphere.size() * sizeof(*d_hittable),
                         hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_accColor, accColor, bufferSize * sizeof(*d_accColor),
						 hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_material, scene.material.data(), 
						 scene.material.size() * sizeof(*d_material),
						 hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_camera, &camera,
		sizeof(*d_camera),
		hipMemcpyHostToDevice))


	if (camera.Moved() || !settings.accumulate)
		accN = 1;
	else
		accN++;

	perPixel << < gridDim, blockDim >> > (
		d_buffer,
		d_accColor,
		imgDim, d_curandState,
		d_hittable,
		scene.sphere.size(),
		d_material,
		d_camera,
		accN);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTimeMs, start, stop);
	gpuErrChk(hipGetLastError());

    gpuErrChk(hipMemcpy(buffer, d_buffer, bufferSize * sizeof(*d_buffer),
                         hipMemcpyDeviceToHost));
	gpuErrChk(hipMemcpy(accColor, d_accColor, bufferSize * sizeof(*d_accColor),
		hipMemcpyDeviceToHost));

	gpuErrChk(hipFree(d_buffer));
	gpuErrChk(hipFree(d_accColor));
	gpuErrChk(hipFree(d_hittable));
	gpuErrChk(hipFree(d_material));
	gpuErrChk(hipFree(d_curandState));
	gpuErrChk(hipFree(d_camera));
	gpuErrChk(hipGetLastError());
}


  float Kernel::getKernelTimeMs() { return kernelTimeMs; }

  Kernel::~Kernel() {}

  void Kernel::setImgDim(glm::uvec2 imgDim) { this->imgDim = imgDim; }

  void Kernel::setBuffer(uint32_t* buffer, glm::vec4* accColor) {
	  this->buffer = buffer;
	  this->accColor = accColor;
  }
